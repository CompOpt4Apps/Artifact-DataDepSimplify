#include "hip/hip_runtime.h"
// this source is derived from CHILL AST originally from file 'mm.c' as parsed by frontend compiler rose

__global__ void kernel_gpu(float *c[1024], float *a[1024], float *b[1024]) {
  int tx = threadIdx.x;
  int by = blockIdx.y;
  int k;
  int j;
  int bx = blockIdx.x;
  {
    for (j = 0; j <= 1023; j += 1) 
      for (k = 0; k <= 1023; k += 1) 
        c[j][bx] = c[j][bx] + a[k][bx] * b[j][k];
  }
}
#include "mm.h"

void normalMM(float c[1024][1024], float a[1024][1024], float b[1024][1024]) {
  float * devRO1ptr;
  float * devRO0ptr;
  float * devRW0ptr;
  hipMalloc((void **)&devRW0ptr, 1048576 * sizeof(float));
  hipMemcpy(devRW0ptr, c, 1048576 * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc((void **)&devRO0ptr, 1048576 * sizeof(float));
  hipMemcpy(devRO0ptr, a, 1048576 * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc((void **)&devRO1ptr, 1048576 * sizeof(float));
  hipMemcpy(devRO1ptr, b, 1048576 * sizeof(float), hipMemcpyHostToDevice);
  dim3 dimGrid0 = dim3(1024, 1);
  dim3 dimBlock0 = dim3(1);
  kernel_gpu<<<dimGrid0,dimBlock0>>>((float (*)[1024])devRW0ptr, (float (*)[1024])devRO0ptr, (float (*)[1024])devRO1ptr);
  hipMemcpy(c, devRW0ptr, 1048576 * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(devRW0ptr);
  hipFree(devRO0ptr);
  hipFree(devRO1ptr);
}
