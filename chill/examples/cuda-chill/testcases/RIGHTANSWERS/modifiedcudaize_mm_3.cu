#include "hip/hip_runtime.h"
// this source is derived from CHILL AST originally from file 'mm.c' as parsed by frontend compiler rose

__global__ void kernel_gpu(float *c[1024], float *a[1024], float *b[1024]) {
  int tx = threadIdx.x;
  int by = blockIdx.y;
  int bx = blockIdx.x;
  int j;
  int i;
  for (i = 0; i <= 1023; i += 1) 
    for (j = 0; j <= 1023; j += 1) 
      {
        c[j][i] = c[j][i] + a[bx][i] * b[j][bx];
      }
}
#include "mm.h"

void normalMM(float c[1024][1024], float a[1024][1024], float b[1024][1024]) {
  float * devRO1ptr;
  float * devRO0ptr;
  float * devRW0ptr;
  hipMalloc((void **)&devRW0ptr, 1048576 * sizeof(float));
  hipMemcpy(devRW0ptr, c, 1048576 * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc((void **)&devRO0ptr, 1048576 * sizeof(float));
  hipMemcpy(devRO0ptr, a, 1048576 * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc((void **)&devRO1ptr, 1048576 * sizeof(float));
  hipMemcpy(devRO1ptr, b, 1048576 * sizeof(float), hipMemcpyHostToDevice);
  dim3 dimGrid0 = dim3(1024, 1);
  dim3 dimBlock0 = dim3(1);
  kernel_gpu<<<dimGrid0,dimBlock0>>>((float (*)[1024])devRW0ptr, (float (*)[1024])devRO0ptr, (float (*)[1024])devRO1ptr);
  hipMemcpy(c, devRW0ptr, 1048576 * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(devRW0ptr);
  hipFree(devRO0ptr);
  hipFree(devRO1ptr);
}
